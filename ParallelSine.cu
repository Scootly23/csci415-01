#include "hip/hip_runtime.h"
//
// Assignment 1: ParallelSine
// CSCI 415: Networking and Parallel Computation
// Spring 2017
// Name(s): Scott St. Amant, Micah Schmit
//
// Sine implementation derived from slides here: http://15418.courses.cs.cmu.edu/spring2016/lecture/basicarch


// standard imports
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>

// problem size (vector length) N
static const int N = 12345678;

// Number of terms to use when approximating sine
static const int TERMS = 6;

//Max number of threads per block
const int threads_per_block = 256;

// kernel function (CPU - Do not modify)
void sine_serial(float *input, float *output)
{
  int i;

  for (i=0; i<N; i++) {
      float value = input[i]; 
      float numer = input[i] * input[i] * input[i]; 
      int denom = 6; // 3! 
      int sign = -1; 
      for (int j=1; j<=TERMS;j++) 
      { 
         value += sign * numer / denom; 
         numer *= input[i] * input[i]; 
         denom *= (2*j+2) * (2*j+3); 
         sign *= -1; 
      } 
      output[i] = value; 
    }
}

// kernel function (CUDA device)
// TODO: Implement your graphics kernel here. See assignment instructions for method information
__global__ void sine_parallel(float *input, float *output) {
  int block_id = blockIdx.x + gridDim.x * blockIdx.y;
	int thread_id = blockDim.x * block_id + threadIdx.x;
  printf("%d\n",thread_id);
   if(thread_id < N){
       float value = input[thread_id]; 
       float numer = input[thread_id] * input[thread_id] * input[thread_id]; 
       int denom = 6; // 3! 
       int sign = -1; 
       for (int j=1; j<=TERMS;j++) 
         { 
          value += sign * numer / denom; 
          numer *= input[thread_id] * input[thread_id]; 
          denom *= (2*j+2) * (2*j+3); 
          sign *= -1; 
         } 
       output[thread_id] = value;
       printf("Output: %d\n",output[thread_id]); 
  }



}
// BEGIN: timing and error checking routines (do not modify)

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}

void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

// END: timing and error checking routines (do not modify)



int main (int argc, char **argv)
{
  //BEGIN: CPU implementation (do not modify)
  float *h_cpu_result = (float*)malloc(N*sizeof(float));
  float *h_input = (float*)malloc(N*sizeof(float));
  //Initialize data on CPU
  int i;
  for (i=0; i<N; i++)
  {
    h_input[i] = 0.1f * i;
  }

  //Execute and time the CPU version
  long long CPU_start_time = start_timer();
  sine_serial(h_input, h_cpu_result);
  long long CPU_time = stop_timer(CPU_start_time, "\nCPU Run Time");
  //END: CPU implementation (do not modify)


  //TODO: Prepare and run your kernel, make sure to copy your results back into h_gpu_result and display your timing results
  float *h_gpu_result = (float*)malloc(N*sizeof(float));

  int vector_size = N * sizeof(float);
	float *d_input, *d_output;
  hipMalloc((void **) &d_input, vector_size);
	hipMalloc((void **) &d_output, vector_size);
	// if (hipMalloc((void **) &d_input, vector_size) != hipSuccess) die("Error allocating GPU memory");
	// if (hipMalloc((void **) &d_output, vector_size) != hipSuccess) die("Error allocating GPU memory");
	
	// Transfer the input vectors to GPU memory
	hipMemcpy(d_input, h_input, vector_size, hipMemcpyHostToDevice);
  int num_blocks = (int) ((float) (N + threads_per_block - 1) / (float) threads_per_block);
	int max_blocks_per_dimension = 65535;
	int num_blocks_y = (int) ((float) (num_blocks + max_blocks_per_dimension - 1) / (float) max_blocks_per_dimension);
	int num_blocks_x = (int) ((float) (num_blocks + num_blocks_y - 1) / (float) num_blocks_y);
	dim3 grid_size(num_blocks_x, num_blocks_y, 1);
  //long long kernal_start_timer = start_timer();
  sine_parallel <<< grid_size, threads_per_block >>> (h_input, h_gpu_result);
  //stop_timer(kernal_start_timer, "\t Kernel execution");

  checkErrors((char*)'k');
  hipMemcpy(h_gpu_result, d_output, vector_size, hipMemcpyDeviceToHost);
  hipFree(d_output);
  hipFree(d_input);

  // Checking to make sure the CPU and GPU results match - Do not modify
  int errorCount = 0;
  for (i=0; i<N; i++)
  {
    if (abs(h_cpu_result[i]-h_gpu_result[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");

  // Cleaning up memory
  free(h_input);
  free(h_cpu_result);
  free(h_gpu_result);
  return 0;
}






